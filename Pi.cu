#include "hip/hip_runtime.h"

#include "hiprand/hiprand_kernel.h"


#include <hiprand/hiprand.h>
#include <iostream>
#include <numeric>

using namespace std;

const long steps = 1 << 21;

__global__ void belongs_circle(double* x, double* y, double* result) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= steps)
		return;
		
    if (((x[tid] - 0.5) * (x[tid] - 0.5)) + ((y[tid] - 0.5) * (y[tid] - 0.5)) <= (0.5 * 0.5)) {
		result[tid] = 1;
	} else {
		result[tid] = 0;
	}
}

int main() {
	const long size = steps * sizeof(double);
	long blockSize = 256;
	long numBlocks = (steps + blockSize - 1) / blockSize;

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

	double *result, *x, *y;
	hipMalloc(&result, size);
	hipMalloc(&x, size);
	hipMalloc(&y, size);
	
	hiprandGenerateUniformDouble(gen, x, steps);
	hiprandGenerateUniformDouble(gen, y, steps);

	belongs_circle <<<numBlocks, blockSize>>>(x, y, result);

	double check[steps];
	hipMemcpy(check, result, size, hipMemcpyDeviceToHost);

	double sum = 0;
	for (long i = 0; i < steps; ++i) {
		sum += check[i];
	}

	cout << "Sum is " << sum << endl;
	cout << "Pi is " << 4 * sum / steps << endl;

	hipFree(result);
	hipFree(x);
	hipFree(y);

	return 0;
}
